#include "hip/hip_runtime.h"
#include "unary_operators.cuh"
#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>
#define ALPHA 1.f
__device__ __forceinline__ float elu_fp32(float x) {
    return x > 0 ? x : ALPHA * (__expf(x) - 1);
}
__global__ void elu_kernel_fp32(float *d_out, const float *d_in, const int N) {
    extern __shared__ float smem[];
    auto block_data = d_in + blockDim.x * blockIdx.x;
    auto tb = cooperative_groups::this_thread_block();
    auto tile = cooperative_groups::tiled_partition<32>(tb);
    auto idx = threadIdx.x;
    smem[idx] = block_data[idx];
    tb.sync();
    smem[threadIdx.x] = elu_fp32(smem[threadIdx.x]);
    tb.sync();
    d_out[blockIdx.x * blockDim.x + threadIdx.x] = smem[threadIdx.x];
}
void elu_fp32_cuda(float *h_out, const float *h_in, const int N) {
    auto nbytes = N * sizeof(float);
    float *d_in, *d_out;
    hipMalloc(reinterpret_cast<void **>(&d_in), nbytes);
    hipMalloc(reinterpret_cast<void **>(&d_out), nbytes);
    hipMemcpy(d_in, h_in, nbytes, hipMemcpyHostToDevice);
    dim3 block = {512, 1, 1};
    dim3 grid = {(N + block.x - 1) / block.x, 1, 1};
    elu_kernel_fp32<<<grid, block, block.x * sizeof(float)>>>(d_out, d_in, N);
    hipMemcpy(h_out, d_out, nbytes, hipMemcpyDeviceToHost);
    hipFree(d_in);
    hipFree(d_out);
}
