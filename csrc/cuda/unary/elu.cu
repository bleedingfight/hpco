#include "unary_ops.cuh"
namespace hpco::unary_ops::cuda {
void elu_fp32_cuda(float *h_out, const float *h_in, const int N) {
    auto nbytes = N * sizeof(float);
    float *d_in, *d_out;
    hipMalloc(reinterpret_cast<void **>(&d_in), nbytes);
    hipMalloc(reinterpret_cast<void **>(&d_out), nbytes);
    hipMemcpy(d_in, h_in, nbytes, hipMemcpyHostToDevice);
    dim3 block = {512, 1, 1};
    dim3 grid = {(N + block.x - 1) / block.x, 1, 1};
    elu_kernel_fp32<<<grid, block, block.x * sizeof(float)>>>(d_out, d_in, N);
    hipMemcpy(h_out, d_out, nbytes, hipMemcpyDeviceToHost);
    hipFree(d_in);
    hipFree(d_out);
}
} // namespace hpco::unary_ops::cuda
